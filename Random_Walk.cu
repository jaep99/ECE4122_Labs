/*
Author: Hyeonjae Park
Class: ECE 4122 (A)
Last Date Modified: Nov 9, 2023
Description:

CUDA-based 2D Random Walk Simulation

*/

#include <iostream>
#include <string>
#include <vector>
#include <ctime>
#include <cstdlib>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <chrono>

#define NUM_BLOCKS 256
#define THREADS_PER_BLOCK 256
#define NUM_POINTS (NUM_BLOCKS * THREADS_PER_BLOCK)


long num_walkers = 1000;  // Number of walkers
long num_steps = 1000;  // Number of steps each walker takes



hipEvent_t start, stop;

// Function for the Random Walk kernel
__global__ void randomWalk(int* resultsX, int* resultsY, int num_walkers, int num_steps, unsigned int seed) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < num_walkers) {
        hiprandState state;
        hiprand_init(seed, tid, 0, &state);  // Use seed
        int x = 0;
        int y = 0;

        for (int step = 0; step < num_steps; step++) {
            float currentState = hiprand_uniform(&state); // Generate a random number
            if (currentState >= 0.0f && currentState < 0.25f) {
                x += 1;
            } else if (currentState >= 0.25f && currentState < 0.5f) {
                x -= 1;
            } else if (currentState >= 0.5f && currentState < 0.75f) {
                y += 1;
            } else {
                y -= 1;
            }
        }
        resultsX[tid] = x;
        resultsY[tid] = y;
    }
}


void FcudaMalloc(int numWalkers, int numSteps) {
    // Device
    int *dX, *dY;
    float DistanceTraveled = 0;
    float AverageDistance = 0;
    int *hX, *hY;
    long long execution_time = 0;

    auto start_time = std::chrono::high_resolution_clock::now();
    auto end_time = std::chrono::high_resolution_clock::now();

    // Timer Start
    start_time = std::chrono::high_resolution_clock::now();
    //cudaEventRecord(start);

    // Allocating device memory
    hipMalloc((void**)&dX, numWalkers * sizeof(int));
    hipMalloc((void**)&dY, numWalkers * sizeof(int));
    //malloc((void**)&hX, numWalkers * sizeof(int));
    //malloc((void**)&hY, numWalkers * sizeof(int));
    hX = (int*)malloc(numWalkers * sizeof(int));
    hY = (int*)malloc(numWalkers * sizeof(int));

    // Kernel Execution
    int block_size = 256;
    int grid_size = ((numWalkers + block_size) / block_size);
    randomWalk<<<grid_size, block_size>>>(dX, dY, numWalkers, numSteps, time(NULL));

    // Transfer host -> device
    hipMemcpy(hX, dX, numWalkers * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(hY, dY, numWalkers * sizeof(int), hipMemcpyDeviceToHost);

    // Distance Calculations (SUM)
    for (int i = 0; i < numWalkers; i++) {
        DistanceTraveled += sqrt(hX[i] * hX[i] + hY[i] * hY[i]);
    }
    AverageDistance = DistanceTraveled / numWalkers;

    // Deallocate device memory
    hipFree(dX);
    hipFree(dY);
    free(hX);
    free(hY);

    // Timer Stop
    end_time = std::chrono::high_resolution_clock::now();
    //cudaEventSynchronize(stop);
    
    // Time computation
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time);
    execution_time = duration.count();
    //float milliseconds = 0;
    //cudaEventElapsedTime(&milliseconds, start, stop);

    std::cout << "Normal CUDA Random Walk:" << std::endl;
    std::cout << "    Time to calculate(microsec): " << execution_time << std::endl;
    std::cout << "    Average distance from origin: " << AverageDistance << std::endl;
}

void FcudaMallocHost(int numWalkers, int numSteps) {
    // Device
    int *dX, *dY;
    float DistanceTraveled = 0;
    float AverageDistance = 0;
    int *hX, *hY;
    long long execution_time = 0;

    auto start_time = std::chrono::high_resolution_clock::now();
    auto end_time = std::chrono::high_resolution_clock::now();

    // Timer Start
    start_time = std::chrono::high_resolution_clock::now();
    //cudaEventRecord(start);

    // Allocating device memory
    hipMalloc((void**)&dX, numWalkers * sizeof(int));
    hipMalloc((void**)&dY, numWalkers * sizeof(int));
    hipHostMalloc((void**)&hX, numWalkers * sizeof(int), hipHostMallocDefault);
    hipHostMalloc((void**)&hY, numWalkers * sizeof(int), hipHostMallocDefault);

    // Kernel Execution
    int block_size = 256;
    int grid_size = ((numWalkers + block_size) / block_size);
    randomWalk<<<grid_size, block_size>>>(dX, dY, numWalkers, numSteps, time(NULL));

    // Transfer host -> device
    hipMemcpy(hX, dX, numWalkers * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(hY, dY, numWalkers * sizeof(int), hipMemcpyDeviceToHost);

    // Distance Calculations (SUM)
    for (int i = 0; i < numWalkers; i++) {
        DistanceTraveled += sqrt(hX[i] * hX[i] + hY[i] * hY[i]);
    }
    AverageDistance = DistanceTraveled / numWalkers;

    // Deallocate device memory
    hipFree(dX);
    hipFree(dY);
    hipHostFree(hX);
    hipHostFree(hY);

    // Timer Stop
    end_time = std::chrono::high_resolution_clock::now();
    //cudaEventSynchronize(stop);
    
    // Time computation
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time);
    execution_time = duration.count();
    //float milliseconds = 0;
    //cudaEventElapsedTime(&milliseconds, start, stop);

    std::cout << "Pinned CUDA Random Walk:" << std::endl;
    std::cout << "    Time to calculate(microsec): " << execution_time << std::endl;
    std::cout << "    Average distance from origin: " << AverageDistance << std::endl;
}

void FcudaMallocManaged(int numWalkers, int numSteps) {
    // Device
    int *dX, *dY, *distances;
    float DistanceTraveled = 0;
    float AverageDistance = 0;
    long long execution_time = 0;

    auto start_time = std::chrono::high_resolution_clock::now();
    auto end_time = std::chrono::high_resolution_clock::now();

    // Allocating device memory
    hipMallocManaged((void**)&dX, numWalkers * sizeof(int));
    hipMallocManaged((void**)&dY, numWalkers * sizeof(int));

    // Kernel Execution
    int block_size = 256;
    int grid_size = ((numWalkers + block_size) / block_size);
    randomWalk<<<grid_size, block_size>>>(dX, dY, numWalkers, numSteps, time(NULL));
    hipDeviceSynchronize();

    // Distance Calculations (SUM)
    for (int i = 0; i < numWalkers; i++) {
        DistanceTraveled += sqrt(dX[i] * dX[i] + dY[i] * dY[i]);
    }
    AverageDistance = DistanceTraveled / numWalkers;

    // Deallocate device memory
    hipFree(dX);
    hipFree(dY);

    // Timer Stop
    end_time = std::chrono::high_resolution_clock::now();
    //cudaEventSynchronize(stop);
    
    // Time computation
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time);
    execution_time = duration.count();
    //float milliseconds = 0;
    //cudaEventElapsedTime(&milliseconds, start, stop);

    std::cout << "Managed CUDA Random Walk:" << std::endl;
    std::cout << "    Time to calculate(microsec): " << execution_time << std::endl;
    std::cout << "    Average distance from origin: " << AverageDistance << std::endl;
}

int main(int argc, char* argv[]) {
    // Flags to track options
    bool verbose = false;

    // Iterate through command-line arguments
    for (int i = 1; i < argc; ++i) {
        std::string arg = argv[i];
        if (arg == "--help" || arg == "-h") {
            // Display usage information
            std::cout << "Usage: " << argv[0] << " [options]" << std::endl;
            std::cout << "Options:" << std::endl;
            std::cout << "  --help (-h)      Show this help message" << std::endl;
            std::cout << "  --verbose (-v)   Enable verbose mode" << std::endl;
            std::cout << "  --W <number>     Number of walkers" << std::endl;
            std::cout << "  --I <number>     Number of steps" << std::endl;
            return 0;
        } else if (arg == "--verbose" || arg == "-v") {
            // Set the verbose flag
            verbose = true;
        } else if (arg == "-W" && i + 1 < argc) {
            // Read the next argument as the number of walkers
            num_walkers = std::stoi(argv[++i]);
        } else if (arg == "-I" && i + 1 < argc) {
            // Read the next argument as the number of steps
            num_steps = std::stoi(argv[++i]);
        } else {
            // Handle unrecognized arguments
            std::cerr << "Error: Unrecognized argument '" << arg << "'" << std::endl;
            return 1;
        }
    }


    hipEventCreate(&start);
    hipEventCreate(&stop);

    FcudaMalloc(num_walkers, num_steps);
    FcudaMallocHost(num_walkers, num_steps);
    FcudaMallocHost(num_walkers, num_steps);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    
    std::cout << "Bye" << std::endl;
    return 0;
}
